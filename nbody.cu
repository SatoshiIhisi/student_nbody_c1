#include "hip/hip_runtime.h"
#include <chrono>
#include <cstdio>
#include <cstdlib>
#include <cmath>
#include <new>

#include <hiprand/hiprand_kernel.h>

#include "cuda_helper.h"
#include "nbody.h"
#include "rendering.h"

// Simulation parameters.
static const int kSeed = 42;
static const float kTimeInterval = 0.5;
static const int kBenchmarkIterations = 10000;

// Physical constants.
static const float kGravityConstant = 6.673e-11;   // gravitational constant

// Array containing all Body objects on device.
__device__ Body bodies[kNumBodies];


__device__ Body::Body(float pos_x, float pos_y,
                      float vel_x, float vel_y, float mass) {
  /* TODO */
}


__device__ void Body::compute_force() {
  /* TODO */
}


__device__ void Body::update(float dt) {
  /* TODO */

  // Bodies should bounce off the wall when they go out of range.
  // Range: [-1, -1] to [1, 1]
}


__global__ void kernel_initialize_bodies() {
  for (int i = threadIdx.x + blockDim.x * blockIdx.x;
       i < kNumBodies; i += blockDim.x * gridDim.x) {
    // Initialize random state.
    hiprandState rand_state;
    hiprand_init(kSeed, i, 0, &rand_state);

    // Create new Body object with placement-new.
    new(bodies + i) Body(/*pos_x=*/ 2 * hiprand_uniform(&rand_state) - 1,
                         /*pos_y=*/ 2 * hiprand_uniform(&rand_state) - 1,
                         /*vel_x=*/ (hiprand_uniform(&rand_state) - 0.5) / 1000,
                         /*vel_y=*/ (hiprand_uniform(&rand_state) - 0.5) / 1000,
                         /*mass=*/ (hiprand_uniform(&rand_state)/2 + 0.5)
                                       * kMaxMass);
  }
}


__global__ void kernel_compute_force() {
  for (int i = threadIdx.x + blockDim.x * blockIdx.x;
       i < kNumBodies; i += blockDim.x * gridDim.x) {
    bodies[i].compute_force();
  }
}


__global__ void kernel_update() {
  /* TODO */
}


// Compute one step of the simulation.
void step_simulation() {
  // n-body consists of 2 CUDA kernels.
  // The first kernel computes the total accumulated gravitational force for
  // every body. The second kernel updates every body's velocity and position.

  kernel_compute_force<<<128, 128>>>();
  gpuErrchk(hipDeviceSynchronize());

  kernel_update<<<128, 128>>>();
  gpuErrchk(hipDeviceSynchronize());
}


void run_interactive() {
  init_renderer();

  // Container for bodies on host.
  Body host_bodies[kNumBodies];

  // Run simulation until user closes the window.
  do {
    // Copy bodies from GPU.
    hipMemcpyFromSymbol(host_bodies, HIP_SYMBOL(bodies), sizeof(Body)*kNumBodies,
                         0, hipMemcpyDeviceToHost);

    // Compute one step.
    step_simulation();
  } while (draw(host_bodies));

  close_renderer();  
}

void run_benchmark() {
  auto time_start = std::chrono::system_clock::now();

  for (int i = 0; i < kBenchmarkIterations; ++i) {
    step_simulation();
  }

  auto time_end = std::chrono::system_clock::now();
  auto elapsed = time_end - time_start;
  auto millis = std::chrono::duration_cast<std::chrono::milliseconds>(elapsed)
      .count();

  printf("Time: %lu ms\n", millis);
}

int main(int argc, char** argv) {
  if (argc != 2) {
    printf("Usage: %s mode\n\nmode 0: Interactive mode\nmode 1: Benchmark\n",
           argv[0]);
    return 1;
  }

  int mode = atoi(argv[1]);

  // Create Body objects.
  kernel_initialize_bodies<<<128, 128>>>();
  gpuErrchk(hipDeviceSynchronize());

  if (mode == 0) {
    run_interactive();
  } else if (mode == 1) {
    run_benchmark();
  } else {
    printf("Invalid mode.\n");
    return 1;
  }

  return 0;
}
